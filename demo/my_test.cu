
#include <hip/hip_runtime.h>
#include<stdio.h>

#define T 512
#define B 10
#define C 16

__global__
void cal(int **W, int ***K, int ***x){
    int block = blockIdx.x;
    int index = threadIdx.x;
    int b = block / C, c = block % C;
    // printf("block is %d; thread is %d K is %d\n",block,index,K[b][c][index]);
    x[b][c][index] = 0;
    for(int l=0;l<index;l++){
        x[b][c][index] += W[c][l] * K[b][c][l];
    }
    return ;
}

int main(){
    // int W[C][T], K[B][C][T], x[B][C][T];
    int **W, ***K, ***x;
    hipMallocManaged(&W, C * sizeof(int*));
    // init W
    for(int i = 0; i < C; i++){
        // allocate unified memoty
        hipMallocManaged((W+i),T * sizeof(int));
        for(int j = 0; j < T; j++){
            W[i][j] = i * T + j;
        }
    }
    hipMallocManaged(&K,B * sizeof(int**));
    hipMallocManaged(&x,B * sizeof(int**));
    // init K
    for(int i = 0; i < B; i++){
        hipMallocManaged((K+i),C * sizeof(int*));
        hipMallocManaged((x+i),C * sizeof(int*));
        for(int j = 0; j < C; j++){
            // allocate unified memoty
            hipMallocManaged((*(x+i)+j), T * sizeof(int));
            hipMallocManaged((*(K+i)+j), T * sizeof(int));
            for(int k = 0; k < T; k++){
                K[i][j][k] = i * (T * C) + j * T + k;
            }
        }
    }
    // calculate

    dim3 num_Blocks(B*C);
    dim3 num_threads(T);
    
    cal<<<num_Blocks,num_threads>>>(W,K,x);
    hipDeviceSynchronize();
    
    // free memory
    for(int i=0;i<B;i++){
        for(int j=0;j<C;j++){
            hipFree(x[i][j]);
            hipFree(K[i][j]);
        }
    }
    for(int i=0;i<B;i++){
        hipFree(x[i]);
        hipFree(K[i]);
    }
    for(int i=0;i<C;i++) hipFree(W[i]);
    hipFree(x);
    hipFree(K);
    hipFree(W);
    
    return 0;
}