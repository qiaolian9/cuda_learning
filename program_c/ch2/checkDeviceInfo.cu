#include<hip/hip_runtime.h>
#include<stdio.h>

int main(int argc, char **argv){
    // device count
    printf("%s starting...\n",argv[0]);
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);

    if(error_id != hipSuccess){
        printf("hipGetDeviceCount returned %d\n -> %s\n",
                (int)error_id,hipGetErrorString(error_id));
        exit(EXIT_FAILURE);
    }

    if(deviceCount == 0){
        printf("There are no available devices that support CUDA\n");
    }else{
        printf("Detected %d CUDA Capable devices\n",deviceCount);
    }

    // device Info (eg device name) 
    int dev = 3;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("Device  %d : %s\n",dev,deviceProp.name);

    return 0;
}