
#include <hip/hip_runtime.h>
#include<iostream>
#include<sys/time.h>
#include<stdlib.h>
__global__
void sumArray(float *A, float *B, float *C, const int N){
    int index = threadIdx.x;
    int block = blockIdx.x;
    int i = block * blockDim.x + index;
    if(i<N) C[i] = A[i] + B[i];
    return ;
}

void initialData(float *ip, const int N){
    time_t t;
    srand((unsigned int) time(&t));

    for(int i=0; i<N; i++){
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
    return ;

}

int main(){
    const int N = 1<<20;
    float *A = (float*)malloc(N * sizeof(float));                  
    float *B = (float*)malloc(N * sizeof(float));
    float *C = (float*)malloc(N * sizeof(float));

    // initial array 
    initialData(A,N);
    initialData(B,N);

    float *d_A, *d_B, *d_C;
    int size = N * sizeof(float);
    hipMalloc((void**)&d_A,size);
    hipMalloc((void**)&d_B,size);
    hipMalloc((void**)&d_C,size);

    // memory copy
    hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);

    // init CUDA
    dim3 num_threads(256);
    dim3 num_Blocks((N + 255) / 256);
    // func_Ptr
    void (*func)(float*, float*, float*, const int);
    func = sumArray;
    
    func<<<num_Blocks,num_threads>>>(d_A,d_B,d_C,N);

    hipMemcpy(C,d_C,size,hipMemcpyDeviceToHost);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(C);

    return 0;
}