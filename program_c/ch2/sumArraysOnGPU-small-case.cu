#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>

#define CHECK(call)                                                                      \
{                                                                                        \
    const hipError_t error = call;                                                      \
    if (error != hipSuccess)                                                            \
    {                                                                                    \
        printf("Error: %s:%d \n",__FILE__,__LINE__);                                     \
        printf("code:%d, reason %s\n",error, hipGetErrorString(error));                 \
        exit(1);                                                                         \
    }                                                                                    \
}                                                                                        \

void checkResults(float *hostRef, float *gpuRef, const int N){
    double epsilon = 1.0E-8;
    bool match = true;
    for(int i=0;i<N;i++){
        if(abs(hostRef[i] - gpuRef[i]) > epsilon){
            match = 0;
            printf("dont match\n");
            printf("%5.2f,%5.2f, index at %d\n",hostRef[i],gpuRef[i],i);
            break;
        }
    }
    if(match) printf("match\n");
}

void initialData(float *ip, int size){
    time_t t;
    srand((unsigned) time(&t));
    for(int i=0;i<size;i++){
        ip[i] = (float)(rand() &0xFF) / 10.0f;
    }
    return ;
}

void sumArray(float *A, float *B, float *C, const int N){
    for(int i=0;i<N;i++){
        C[i] = A[i] + B[i];
    }
    return ;
}

__global__
void sumArrayOnGPU(float *A, float *B, float *C){
    int index = threadIdx.x;
    int block = blockIdx.x;
    int i = block * blockDim.x + index;
    C[i] = A[i] + B[i];
    return ;
}

int main(){
    printf("Starting ...\n");
    int dev = 4;
    hipSetDevice(dev);
    const int N = 1<<25;
    int bytesize = N * sizeof(float);
    
    // init cpu data
    float *A, *B, *hostRef, *gpuRef;
    A = (float*)malloc(bytesize);
    B = (float*)malloc(bytesize);
    hostRef = (float*)malloc(bytesize);
    gpuRef = (float*)malloc(bytesize);

    initialData(A,N);
    initialData(B,N);
    memset(hostRef,0,bytesize);
    memset(gpuRef,0,bytesize);
    
    // init gpu data
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A,bytesize);
    hipMalloc((void**)&d_B,bytesize);
    hipMalloc((void**)&d_C,bytesize);
    
    hipMemcpy(d_A,A,bytesize,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,bytesize,hipMemcpyHostToDevice);

    // cpu calculate
    sumArray(A,B,hostRef,N);

    // gpu calculate
    dim3 block(256);
    dim3 grid((N + block.x -1) / block.x);
    sumArrayOnGPU<<<grid,block>>>(d_A,d_B,d_C);
    hipDeviceSynchronize();
    hipMemcpy(gpuRef,d_C,bytesize,hipMemcpyDeviceToHost);

    hipDeviceReset();
    // check
    checkResults(hostRef,gpuRef,N);

    // free memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    free(A);
    free(B);
    free(hostRef);
    free(gpuRef);

    return 0;
}