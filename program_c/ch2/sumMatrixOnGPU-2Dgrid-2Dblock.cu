#include "hip/hip_runtime.h"
#include<hip/hip_runtime.h>
#include<stdio.h>
#include<sys/time.h>


#define CHECK(call)                                                                      \
{                                                                                        \
    const hipError_t error = call;                                                        \
    if (error != hipSuccess)                                                            \
    {                                                                                    \
        printf("Error: %s:%d \n",__FILE__,__LINE__);                                     \
        printf("code:%d, reason %s\n",error, hipGetErrorString(error));                 \
        exit(1);                                                                         \
    }                                                                                    \
}                                                                                        \

void initialData(float *ip, const int nxy){
    time_t t;
    srand((unsigned) time(&t));
    for(int i=0;i<nxy;i++){
        ip[i] = (float)(rand() & 0xFF) / 10.0f;
    }
}                                                                                     

void checkResults(float *hostRef, float *gpuRef, const int nx, const int ny){
    bool match = 1;
    double epsilon = 1.0E-9;
    for(int i=0;i<ny;i++){
        for(int j=0;j<nx;j++){
            int row = i * nx;
            if(abs((hostRef + row)[j] - (gpuRef + row)[j]) > epsilon){
                match = 0;
                printf("dont match!!! Error index (%d %d), host %5.2f, device %5.2f\n"  \
                        ,i,j,(hostRef + row)[j],(gpuRef + row)[j]);
                break;
            }
        }
        if(!match) break;
    }
    if(match) printf("Matrix Match!\n");
}

double cpuMSecond(){
    struct timeval tp;
    gettimeofday(&tp,NULL);
    return ((double)tp.tv_sec * 1.0E3 + (double)tp.tv_usec * 1.0E-3);
}

void sumMatrix(float *A, float *B, float *hostRef, const int nx, const int ny){
    for(int i=0;i<ny;i++){
        for(int j=0;j<nx;j++){
            (hostRef + i * nx)[j] = (B + i * nx)[j] + (A + i * nx)[j];
        }
    }
}

__global__
void sumMatrixOnGPU(float *d_A, float *d_B, float *d_C, const int nx, const int ny){
    int ix = threadIdx.x + blockIdx.x * blockDim.x;
    int iy = threadIdx.y + blockIdx.y * blockDim.y;
    int index = iy * nx + ix;
    if(ix < nx && iy < ny){
        d_C[index] = d_A[index] + d_B[index];
    }
    return ;
}

int main(int argc, char **argv){
    // initial environment
    double iStart, iElaps;
    printf("%s starting...\n",argv[0]);
    int dev = 3;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp,dev));
    printf("Using Device %d : %s\n",dev,deviceProp.name);
    CHECK(hipSetDevice(dev));

    // initial data
    int n;
    scanf("%d",&n);
    int nx = 1 << n;
    int ny = 1 << n;
    int nxy = nx * ny;
    int nBytes = nxy * sizeof(float);

    // initial host memory
    float *A, *B, *hostRef, *gpuRef;
    A = (float*)malloc(nBytes);
    B = (float*)malloc(nBytes);
    hostRef = (float*)malloc(nBytes);
    gpuRef = (float*)malloc(nBytes);
    initialData(A,nxy);
    initialData(B,nxy);

    // initial device memory
    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, nBytes);
    hipMalloc((void**)&d_B, nBytes);
    hipMalloc((void**)&d_C, nBytes);

    hipMemcpy(d_A,A,nBytes,hipMemcpyHostToDevice);
    hipMemcpy(d_B,B,nBytes,hipMemcpyHostToDevice);

    // host code run
    iStart = cpuMSecond();
    sumMatrix(A,B,hostRef,nx,ny);
    iElaps = cpuMSecond() - iStart;
    printf("Matrix add time cost time %f ms\n",iElaps);

    // device code run
    dim3 block(32,32);
    dim3 grid((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);
    
    iStart = cpuMSecond();
    sumMatrixOnGPU<<<grid,block>>>(d_A,d_B,d_C,nx,ny);
    hipDeviceSynchronize();
    iElaps = cpuMSecond() - iStart;
    printf("Matrix add cuda(2Dgrid-2Dblock) time cost %f ms\n",iElaps);

    hipMemcpy(gpuRef,d_C,nBytes,hipMemcpyDeviceToHost);
    checkResults(hostRef,gpuRef,nx,ny);

    // free memory
    free(A);
    free(B);
    free(hostRef);
    free(gpuRef);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipDeviceReset();

    return 0;
}