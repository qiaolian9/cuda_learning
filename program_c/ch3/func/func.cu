#include<hip/hip_runtime.h>
#include<stdio.h>
#include "metric.h"

int func(void (*p)(int *g_idata, int *g_odata, unsigned int n), int *g_idata, int *g_odata,
        int *h_idata, int *h_odata, unsigned int n, size_t nBytes, dim3 block, dim3 grid, const char* s){
    double iStart, iElaps;
    hipMemcpy(g_idata,h_idata,nBytes,hipMemcpyHostToDevice);
    iStart = cpuMSecond();
    (*p)<<<grid,block>>>(g_idata,g_odata,n);
    hipDeviceSynchronize();
    hipMemcpy(h_odata,g_odata,grid.x * sizeof(int),hipMemcpyDeviceToHost);
    int gpu_sum = 0;
    for(int i=0;i<grid.x;i++){
        gpu_sum += h_odata[i];
    }
    iElaps = cpuMSecond() - iStart;
    printf("%s cuda time cost %f ms & ",s,iElaps);
    printf("func<<<(%d %d):(%d %d)>>>\n",grid.x,grid.y,block.x,block.y);
    return gpu_sum;
}