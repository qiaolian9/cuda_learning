#include<stdio.h>
#include<hip/hip_runtime.h>
#define M(x,n){x = (float*)malloc(n);}
#define cM(x,n){hipMalloc((void**)&x,n);}

int main(int argc,char **argv){
    int dev = 3;
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp,dev);
    printf("%s starting...\n",argv[0]);
    printf("Using Device %d : %s\n",dev,deviceProp.name);
    hipSetDevice(dev);

    // initial data
    unsigned int n = 1<<22;
    unsigned int nBytes = n * sizeof(float);
    printf("Data size %5.2f MB\n",nBytes / (1024.0f * 1024.0f));

    // allocate the host memory
    float *h_g, *d_g;
    M(h_g,nBytes);
    cM(d_g,nBytes);

    for(unsigned int i=0;i<n;i++) h_g[i] = 1.0f;

    // transfer data
    hipMemcpy(d_g,h_g,nBytes,hipMemcpyHostToDevice);
    hipMemcpy(h_g,d_g,nBytes,hipMemcpyDeviceToHost);
    
    // free memory
    free(h_g);
    hipFree(d_g);
    hipDeviceReset();
    return 0;
}